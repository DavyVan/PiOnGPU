#include <hip/hip_runtime.h>
#include <>

#include "functions.h"


__device__ __int64 biexp(__int64 a, __int64 b, __int64 mod)
{
	__int64 ret = 1;
	while (b != 0)
	{
		if (b % 2)
			ret = (ret * a) % mod;
		a = (a * a) % mod;
		b /= 2;
	}
	return ret;
}

char hex_table[] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'A', 'B', 'C', 'D', 'E', 'F' };

__device__ void DecToHexSingle(unsigned char dec, char * hex)
{
	*hex = hex_table[dec >> 4];
	*(hex + 1) = hex_table[dec & 0x0F];
}

__device__ void DecToHexArray(unsigned char dec[], char hex[], int count)
{
	for (int i = 0; i < count; i++)
		DecToHexSingle(dec[i], &hex[2 * i]);
}

__device__ void DecToHexModDiv(unsigned char dec, char &hex)
{
	hex = hex_table[dec];
}

__device__ double _16dsj(__int64 d, int j)
{
	double sum = 0;
	for (__int64 k = 0; k <= d; k++)
	{
		sum += (double) biexp(16, d - k, 8 * k + j) / (8 * k + j);
	}

	return sum - (int)sum;
}