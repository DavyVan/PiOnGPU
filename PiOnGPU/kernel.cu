/*
* Copyright for DeviceQuery 
*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* PiOnGPU computes Pi on GPU for 2,000,000 Hex
*/

#include<memory>
#include<iostream>

#include<hip/hip_runtime.h>
#include"CUDA Helper\hip/hip_runtime_api.h"

void errorPrint(hipError_t error_id)
{
	printf("ERROR: %d - %s\n", (int) error_id, hipGetErrorString(error_id));
	//printf("Error occured, terminating...\n");
	return;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index - 1].Cores);
	return nGpuArchCoresPerSM[index - 1].Cores;
}

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	pArgc = &argc;
	pArgv = argv;

	printf("Starting...\n");

	/**********************************DeviceQuery*****************************************/
	printf("Device checking...\n");
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess)
	{
		errorPrint(error_id);
		printf("Device check failed, terminating...\n");
		exit(EXIT_FAILURE);
	}

	if (deviceCount == 0)
	{
		printf("ERROR: No CUDA device available\n");
		printf("Device check failed, terminating...\n");
		exit(EXIT_FAILURE);
	}
	else
		printf("\nDetected %d CUDA Capable device(s)\n\n", deviceCount);

	int deviceIterator, driverVersion, runtimeVersion;
	for (deviceIterator = 0; deviceIterator < deviceCount; deviceIterator++)
	{
		hipSetDevice(deviceIterator);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, deviceIterator);

		printf("Device %d: %s\n", deviceIterator, deviceProp.name);

		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", 
			driverVersion / 1000, 
			(driverVersion % 100) / 10, 
			runtimeVersion / 1000, 
			(runtimeVersion % 100) / 10);
		printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
			(float) deviceProp.totalGlobalMem / 1048576.0f,
			(unsigned long long) deviceProp.totalGlobalMem);

		printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
			deviceProp.multiProcessorCount,
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
		printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

#if CUDART_VERSION >= 5000
		// This is supported in CUDA 5.0 (runtime API device properties)
		printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
		printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);

		if (deviceProp.l2CacheSize)
		{
			printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
		}
#endif
		printf("  Total amount of constant memory:               %lu bytes\n", deviceProp.totalConstMem);
		printf("  Total amount of shared memory per block:       %lu bytes\n", deviceProp.sharedMemPerBlock);
		printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
		printf("  Max dimension size of a thread block (x,y,z):  (%d, %d, %d)\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("  Max dimension size of a grid size    (x,y,z):  (%d, %d, %d)\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", 
			(deviceProp.deviceOverlap ? "Yes" : "No"), 
			deviceProp.asyncEngineCount);
		printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
		printf("  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
		printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
		printf("  Warp Size:                                     %d\n", deviceProp.warpSize);
	}
	printf("Device check done...\n");
	//Reset before computing Pi
	hipDeviceReset();
	getchar();
	exit(EXIT_SUCCESS);
}